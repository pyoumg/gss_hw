#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>




#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1
#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

// CUDA event ��ü�� ����Ͽ� Ŀ�� ����ð� ����
void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;




#define N_EQUATIONS	(1<<20)
#define BLOCK_SIZE 256
float(*pX0), (*pX1);
float(*pX0_G), (*pX1_G);//gpu
float(*pFX0), (*pFX1);//cpu
float(*pFX0_G), (*pFX1_G);//gpu
float(*pA), (*pB), (*pC);


void init_data(void) {
	srand((unsigned)time(NULL));
	FILE* fp_a = fopen("A.bin", "wb"); //write binary
	FILE* fp_b = fopen("B.bin", "wb");
	FILE* fp_c = fopen("C.bin", "wb");

	int i, j;
	float x;

	for (i = 0; i < N_EQUATIONS; i++) {
		x = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
		fwrite(&x, sizeof(float), 1, fp_a);

	}

	for (i = 0; i < N_EQUATIONS; i++) {
		x = 1.0f * ((float)rand() / RAND_MAX)  +7.0f;
		fwrite(&x, sizeof(float), 1, fp_b);

	}
	for (i = 0; i < N_EQUATIONS; i++) {
		x = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
		fwrite(&x, sizeof(float), 1, fp_c);

	}

	fclose(fp_a);
	fclose(fp_b);
	fclose(fp_c);
	return;
}

void init_Mat(void)
{
	srand((unsigned)time(NULL));
	FILE* fp_a = fopen("A.bin", "rb");
	FILE* fp_b = fopen("B.bin", "rb");
	FILE* fp_c = fopen("C.bin", "rb");


	pFX0 = new float[N_EQUATIONS];
	pFX1 = new float[N_EQUATIONS];
	pFX0_G = new float[N_EQUATIONS];
	pFX1_G = new float[N_EQUATIONS];

	pX0 = new float[N_EQUATIONS];
	pX1 = new float[N_EQUATIONS];
	pX0_G = new float[N_EQUATIONS];
	pX1_G = new float[N_EQUATIONS];

	pA = new float[N_EQUATIONS];
	pB = new float[N_EQUATIONS];
	pC = new float[N_EQUATIONS];


	fread(pA, sizeof(float), N_EQUATIONS, fp_a);
	fread(pB, sizeof(float), N_EQUATIONS, fp_b);
	fread(pC, sizeof(float), N_EQUATIONS, fp_c);

	fclose(fp_a);
	fclose(fp_b);
	fclose(fp_c);
}

void write_data() {
	FILE* fp_x0 = fopen("X0.bin", "wb");
	FILE* fp_x1 = fopen("X1.bin", "wb");
	FILE* fp_fx0 = fopen("FX0.bin", "wb");
	FILE* fp_fx1 = fopen("FX1.bin", "wb");

	for (int i = 0; i < N_EQUATIONS; i++) {
		fwrite(&pX0_G[i], sizeof(float), 1, fp_x0);
		fwrite(&pX1_G[i], sizeof(float), 1, fp_x1);
		fwrite(&pFX0_G[i], sizeof(float), 1, fp_fx0);
		fwrite(&pFX1_G[i], sizeof(float), 1, fp_fx1);
	}


	fclose(fp_x0);
	fclose(fp_x1);
	fclose(fp_fx0);
	fclose(fp_fx1);
}

void find_roots_CPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n) {
	int i;
	float a, b, c, d, x0, x1, tmp, temp;
	for (i = 0; i < n; i++) {
		a = A[i]; b = B[i]; c = C[i];
		d = sqrtf(b * b - 4.0f * a * c);
		tmp = 1.0f / (2.0f * a);
		X0[i] = x0 = (-b - d) * tmp;
		X1[i] = x1 = (-b + d) * tmp;

		if (x0 > x1) {
			temp = x0;
			x0 = x1;
			x1 = temp;
			X0[i] = x0, X1[i] = x1;
		}


		FX0[i] = (a * x0 + b) * x0 + c;
		FX1[i] = (a * x1 + b) * x1 + c;

		//printf("%f %f %f %f\n", X0[i], X1[i], FX0[i], FX1[i]);
	}
}

__global__ void find_roots_Kernel(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	float a, b, c, d, x0, x1, tmp, temp;

	a = A[id]; b = B[id]; c = C[id];
	d = sqrtf(b * b - 4.0f * a * c);
	tmp = 1.0f / (2.0f * a);
	X0[id] = x0 = (-b - d) * tmp;
	X1[id] = x1 = (-b + d) * tmp;
	if (x0 > x1) {
		temp = x0;
		x0 = x1;
		x1 = temp;
		X0[id] = x0, X1[id] = x1;
	}
	FX0[id] = (a * x0 + b) * x0 + c;
	FX1[id] = (a * x1 + b) * x1 + c;

}


hipError_t find_roots_GPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n);


int main()
{
	int n_elements;

	srand((unsigned int)time(NULL));
	n_elements = N_EQUATIONS;

	printf("*** Data size : %d\n\n", n_elements);

	init_data();
	init_Mat();
	CHECK_TIME_START;
	find_roots_CPU(pA, pB, pC, pX0, pX1, pFX0, pFX1, n_elements);
	CHECK_TIME_END(compute_time);

	printf("***CPU X0[8] = %f Time taken = %.6fms\n",pX0[8],compute_time);


	hipError_t cudaStatus = find_roots_GPU(pA, pB, pC, pX0_G, pX1_G, pFX0_G, pFX1_G, n_elements);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "find_roots_GPU failed!");
		return 1;
	}
	printf("***GPU X0[8] = %f Time taken = %.6fms\n", pX0_G[8], device_time);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	write_data();
	return 0;
}
hipError_t find_roots_GPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n) {
	float * pa, * pb, * pc, * x0, * x1, * fx0, * fx1;

	float size = n * sizeof(float);

	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL

	hipDeviceProp_t deviceProp;
	CUDA_CALL(hipGetDeviceProperties(&deviceProp, 0));

		CUDA_CALL(hipMalloc(&pa, size))
			CUDA_CALL(hipMemcpy(pa, A, size, hipMemcpyHostToDevice))
		CUDA_CALL(hipMalloc(&pb, size))
			CUDA_CALL(hipMemcpy(pb, B, size, hipMemcpyHostToDevice))
		CUDA_CALL(hipMalloc(&pc, size))
			CUDA_CALL(hipMemcpy(pc, C, size, hipMemcpyHostToDevice))

		CUDA_CALL(hipMalloc(&x0, size))
		CUDA_CALL(hipMalloc(&x1, size))
		CUDA_CALL(hipMalloc(&fx0, size))
		CUDA_CALL(hipMalloc(&fx1, size))

		

		// Assume that width and height are multiples of BLOCK SIZE.
		dim3 dimBlock(BLOCK_SIZE,1);
	dim3 dimGrid(n / dimBlock.x,1);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();
	find_roots_Kernel << < dimGrid, dimBlock >> > (pa, pb, pc, x0, x1, fx0, fx1);
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();
	CUDA_CALL(hipGetLastError())

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())

		CUDA_CALL(hipMemcpy(X0, x0, size, hipMemcpyDeviceToHost))
		CUDA_CALL(hipMemcpy(X1, x1, size, hipMemcpyDeviceToHost))
		CUDA_CALL(hipMemcpy(FX0, fx0, size, hipMemcpyDeviceToHost))
		CUDA_CALL(hipMemcpy(FX1, fx1, size, hipMemcpyDeviceToHost))

		Error:
	hipFree(pa);
	hipFree(pb);
	hipFree(pc);
	hipFree(x0);
	hipFree(x1);
	hipFree(fx0);
	hipFree(fx1);
	return cudaStatus;
}
