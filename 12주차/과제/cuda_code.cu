#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2
#define BLOCK_SIZE_X  8
#define  BLOCK_SIZE_Y  8
#define SHARED_MEM(x, y)		sharedBuffer[SMW * (y) + (x)]
__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {	
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int id = width * row + col;

	int r, c,kernel_idx=0;
	int r_idx, c_idx;
	char temp = 0;

	for (r = -2; r <= 2; r++) {
		r_idx = row + r;
		if (r_idx < 0) r_idx = 0;
		else if (r_idx > height - 1) r_idx = height - 1;
		for (c = -2; c <= 2; c++) {
			c_idx = col+ c;
			if (c_idx < 0) c_idx = 0;
			else if (c_idx > width - 1) c_idx = width - 1;
			temp += constant_gaussian_kernel[kernel_idx++] * d_bitmaps[r_idx *width+ c_idx];

		}
	}
	d_Gaussian[id] = temp;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int id = width * row + col;
	int SMW = blockDim.x + 4;//2*2



	int r, c, kernel_idx = 0;
	int2 input_idx;
	int idx = 0;
	int loc_column=threadIdx.x,loc_row=threadIdx.y;// index
	char temp = 0;

	int side_left=0, side_right = 0;
	

	SHARED_MEM(loc_column+2, loc_row+2 ) = d_bitmaps[id];


	if (blockIdx.x == 0 || blockIdx.x == gridDim.x - 1 || blockIdx.y == 0 || blockIdx.y == gridDim.y - 1) {
		if (loc_column < 2) {
			input_idx.x = (col - 2 < 0) ? 0 : col - 2;
			SHARED_MEM(loc_column, loc_row + 2) = d_bitmaps[row*width+input_idx.x];
			side_left = 1;
		}
		else if (loc_column >= blockDim.x - 2) {
			input_idx.x = (col + 2 >= width) ? width - 1 : col + 2;
			SHARED_MEM(loc_column + 2 * 2, loc_row + 2) = d_bitmaps[row * width + input_idx.x]; 
			side_right = 1;
		}

		if (loc_row < 2) {
			input_idx.y = (row - 2 < 0) ? 0 : row - 2;
			SHARED_MEM(loc_column + 2, loc_row) = d_bitmaps[input_idx.y * width + col]; 
			if (side_left == 1) {
				input_idx.x = (col - 2 < 0) ? 0 : col - 2;
				input_idx.y = (row - 2 < 0) ? 0 : row - 2;
				SHARED_MEM(loc_column, loc_row) = d_bitmaps[input_idx.y * width + input_idx.x]; 
			}
			if (side_right == 1) {
				input_idx.x = (col + 2 >= width) ? width - 1 : col + 2;
				input_idx.y = (row - 2 < 0) ? 0 : row - 2;
				SHARED_MEM(loc_column + 2 * 2, loc_row) = d_bitmaps[input_idx.y * width + input_idx.x]; 
			}
		}
		else if (loc_row >= blockDim.y - 2) {
			input_idx.y = (row + 2 >= height) ? height - 1 : row + 2;
			SHARED_MEM(loc_column + 2, loc_row + 2 * 2) = d_bitmaps[input_idx.y * width + col];
			if (side_left == 1) {
				input_idx.x = (col - 2 < 0) ? 0 : col - 2;
				input_idx.y = (row + 2 >= height) ? height - 1 : row + 2;
				SHARED_MEM(loc_column, loc_row + 2 * 2) = d_bitmaps[input_idx.y * width + input_idx.x]; 
			}
			if (side_right == 1) {
				input_idx.x = (col + 2 >= width) ? width - 1 : col + 2;
				input_idx.y = (row + 2 >= height) ? height - 1 : row + 2;
				SHARED_MEM(loc_column + 2 * 2, loc_row + 2 * 2) = d_bitmaps[input_idx.y * width + input_idx.x]; 
			}
		}
	}
	else {
		if (loc_column < 2) {
			SHARED_MEM(loc_column, loc_row + 2) = d_bitmaps[row * width + col - 2]; 
			side_left = 1;
		}
		else if (loc_column >= blockDim.x - 2) {
			SHARED_MEM(loc_column + 2 * 2, loc_row + 2) = d_bitmaps[row * width + col + 2]; 
			side_right = 1;
		}

		if (loc_row < 2) {
			SHARED_MEM(loc_column + 2, loc_row) = d_bitmaps[row * width + col - 2];
			if (side_left == 1)
				SHARED_MEM(loc_column, loc_row) = d_bitmaps[(row-2) * width + col - 2]; 
			if (side_right == 1)
				SHARED_MEM(loc_column + 2 * 2, loc_row) = d_bitmaps[(row - 2) * width + col + 2]; 
		}
		else if (loc_row >= blockDim.y - 2) {
			SHARED_MEM(loc_column + 2, loc_row + 2 * 2) = d_bitmaps[(row + 2) * width + col ]; 
			if (side_left == 1)
				SHARED_MEM(loc_column, loc_row + 2 * 2) = d_bitmaps[(row + 2) * width + col - 2]; 
			if (side_right == 1)
				SHARED_MEM(loc_column + 2 * 2, loc_row + 2 * 2) = d_bitmaps[(row + 2) * width + col + 2]; 
		}
	}



	__syncthreads();

	for (r = loc_row - 2; r <= loc_row + 2; r++) {
		for (c = loc_column - 2; c <= loc_column + 2; c++) {
			temp+=SHARED_MEM(c + 2, r + 2)* constant_gaussian_kernel[kernel_idx++];
		}
	}
	d_Gaussian[id] = temp;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	Set_Gaussain_Kernel();
	CUDA_CALL(hipSetDevice(0));
	unsigned int total_pixel = width * height;

	unsigned char *d_bitmaps, *d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	
	int gridy = height / blockDim.y;
	if (height % blockDim.y != 0)
		gridy++;
	dim3 gridDim(width / blockDim.x, gridy);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDim >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << < gridDim, blockDim, sizeof(unsigned char) * (blockDim.x + 2 * Window) * (blockDim.y + 2 * Window)>> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}